#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// CUDA Kernel
////////////////////////////////////////////////////////////////////////////////

__constant__ int const_d[CT_MEM_SIZE];

__global__ void foo(int *gid_d)
{
    extern __shared__ int shared_mem[];

	// size of the block
    int blockSize = blockDim.x * blockDim.y; 

    // global thread ID in thread block
    int tidb = (blockDim.x * threadIdx.y + threadIdx.x); 

    // global thread ID in grid
    int tidg = (blockIdx.y * gridDim.x * blockSize + blockIdx.x * blockSize + tidb); 

    shared_mem[tidb] = gid_d[tidg];
    
    __syncthreads();

	/* shared memory */
    shared_mem[tidb] = (tidg + const_d[tidg % CT_MEM_SIZE]);

    __syncthreads();

    gid_d[tidg] = shared_mem[tidb];
}

